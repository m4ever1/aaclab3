#include "hip/hip_runtime.h"
/*****************************************************************************/
/*IMPORTANT:  READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.         */
/*By downloading, copying, installing or using the software you agree        */
/*to this license.  If you do not agree to this license, do not download,    */
/*install, copy or use the software.                                         */
/*                                                                           */
/*                                                                           */
/*Copyright (c) 2005 Northwestern University                                 */
/*All rights reserved.                                                       */

/*Redistribution of the software in source and binary forms,                 */
/*with or without modification, is permitted provided that the               */
/*following conditions are met:                                              */
/*                                                                           */
/*1       Redistributions of source code must retain the above copyright     */
/*        notice, this list of conditions and the following disclaimer.      */
/*                                                                           */
/*2       Redistributions in binary form must reproduce the above copyright   */
/*        notice, this list of conditions and the following disclaimer in the */
/*        documentation and/or other materials provided with the distribution.*/ 
/*                                                                            */
/*3       Neither the name of Northwestern University nor the names of its    */
/*        contributors may be used to endorse or promote products derived     */
/*        from this software without specific prior written permission.       */
/*                                                                            */
/*THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS ``AS    */
/*IS'' AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED      */
/*TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT AND         */
/*FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL          */
/*NORTHWESTERN UNIVERSITY OR ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT,       */
/*INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES          */
/*(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR          */
/*SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)          */
/*HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,         */
/*STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN    */
/*ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE             */
/*POSSIBILITY OF SUCH DAMAGE.                                                 */
/******************************************************************************/
/*************************************************************************/
/**   File:         kmeans_clustering.c                                 **/
/**   Description:  Implementation of regular k-means clustering        **/
/**                 algorithm                                           **/
/**   Author:  Wei-keng Liao                                            **/
/**            ECE Department, Northwestern University                  **/
/**            email: wkliao@ece.northwestern.edu                       **/
/**                                                                     **/
/**   Edited by: Jay Pisharath                                          **/
/**              Northwestern University.                               **/
/**                                                                     **/
/**   ================================================================  **/
/**																		**/
/**   Edited by: Sang-Ha  Lee											**/
/**				 University of Virginia									**/
/**																		**/
/**   Description:	No longer supports fuzzy c-means clustering;	 	**/
/**					only regular k-means clustering.					**/
/**					Simplified for main functionality: regular k-means	**/
/**					clustering.											**/
/**                                                                     **/
/*************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "kmeans.h"
#include <omp.h>

#define RANDOM_MAX 2147483647

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif

extern double wtime(void);

__global__ void find_nearest_point(float  *pt,          /* [nfeatures] */
                       int     nfeatures,
                       float **pts,         /* [npts][nfeatures] */ 
                       int     npts,
                       float*    dists)
{
    //  fp;
    // fp = fopen("./ans_cuda.txt", "a");
    float min_dist=FLT_MAX;
    float* dist;
    // dist = (float*) malloc(sizeof(float)*npts);
    // hipMallocManaged(&dist, sizeof(float));
    /* find the cluster center id with min distance to pt */
    int i = blockIdx.x;
    // if (i<npts) {
        dists[i] = 0;
        euclid_dist_2(pt, pts[i], nfeatures, &(dists[i]));  /* no need square root */
        
        // fprintf(fp, "ans = %f\n", *dist);
        // atomicMax(&(dist[i]), min_dist);
        // atomicExch(&min_dist, dist[i]);
        // *index = 2;
        // if (dist[i] != ) 
        // {
        //     min_dist = dist[j];
        //     *index    = j;
        // }

    // }
    // ;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* multi-dimensional spatial Euclid distance square */
__device__ void euclid_dist_2(float *pt1,
                    float *pt2,
                    int    numdims, float* dist)
{
    int i = threadIdx.x;
    float ans=0.0;
    // printf("i = %f\n", dist);
    for (i=0; i<numdims; i++)
    {
        // atomicAdd(dist, (pt1[i]-pt2[i]) * (pt1[i]-pt2[i]));
        *dist += (pt1[i]-pt2[i]) * (pt1[i]-pt2[i]);
    }
        // float ans += (pt1[i]-pt2[i]) * (pt1[i]-pt2[i]);
    // *dist = ans;

}


/*----< kmeans_clustering() >---------------------------------------------*/
float** kmeans_clustering(float **feature,    /* in: [npoints][nfeatures] */
                          int     nfeatures,
                          int     npoints,
                          int     nclusters,
                          float   threshold,
                          int    *membership) /* out: [npoints] */
{

    int      i, j, n=0, loop=0;
    int     *new_centers_len; /* [nclusters]: no. of points in each cluster */
    float    delta;
    float  **clusters;   /* out: [nclusters][nfeatures] */
    float  **new_centers;     /* [nclusters][nfeatures] */
  

    /* allocate space for returning variable clusters[] */
    hipMallocManaged(&clusters, nclusters *             sizeof(float*));
    hipMallocManaged(clusters, nclusters * nfeatures * sizeof(float));
    // clusters    = (float**) malloc(nclusters *             sizeof(float*));
    // clusters[0] = (float*)  malloc(nclusters * nfeatures * sizeof(float));
    for (i=1; i<nclusters; i++)
        clusters[i] = clusters[i-1] + nfeatures;

    /* randomly pick cluster centers */
    for (i=0; i<nclusters; i++) {
        //n = (int)rand() % npoints;
        for (j=0; j<nfeatures; j++)
            clusters[i][j] = feature[n][j];
		n++;
    }

    for (i=0; i<npoints; i++)
		membership[i] = -1;

    /* need to initialize new_centers_len and new_centers[0] to all 0 */
    new_centers_len = (int*) calloc(nclusters, sizeof(int));

    new_centers    = (float**) malloc(nclusters *            sizeof(float*));
    new_centers[0] = (float*)  calloc(nclusters * nfeatures, sizeof(float));
    for (i=1; i<nclusters; i++)
        new_centers[i] = new_centers[i-1] + nfeatures;
 
    float * dists_device;
    float * dists_host;
    hipMalloc(&dists_device, sizeof(float)*nclusters);
    dists_host = (float*) malloc(sizeof(float)*nclusters);
    int index;
    float min_dist=FLT_MAX;
    FILE* fp = fopen("./index_cuda.txt", "a");
    do {
        
        delta = 0.0;
        
        for (i=0; i<npoints; i++) {
            /* find the index of nestest cluster centers */
            min_dist=FLT_MAX;
	        find_nearest_point<<<nclusters, nfeatures>>>(feature[i], nfeatures, clusters, nclusters, dists_device);
            hipDeviceSynchronize();
            hipMemcpy(dists_host, dists_device, sizeof(float)*nclusters, hipMemcpyDeviceToHost);
            for(int k=0;k<nclusters;k++)
            {
                if (dists_host[k] < min_dist) 
                {
                    min_dist = dists_host[k];
                    index    = k;
                    // printf("%f\n", dists_host[k]);
                }
            }
            fprintf(fp, "index = %d\n", index);
            /* if membership changes, increase delta by 1 */
	        if (membership[i] != index) delta += 1.0;

	        /* assign the membership to object i */
	        membership[i] = index;

	        /* update new cluster centers : sum of objects located within */
	        new_centers_len[index]++;
	        for (j=0; j<nfeatures; j++)          
				new_centers[index][j] += feature[i][j];
        }
      

	/* replace old cluster centers with new_centers */
        for (i=0; i<nclusters; i++) {
            for (j=0; j<nfeatures; j++) {
                if (new_centers_len[i] > 0)
					clusters[i][j] = new_centers[i][j] / new_centers_len[i];
				new_centers[i][j] = 0.0;   /* set back to 0 */
			}
			new_centers_len[i] = 0;   /* set back to 0 */
		}
            
        //delta /= npoints;
    } while (delta > threshold);
    fclose(fp);
    hipFree(dists_device);
    free(dists_host);
    free(new_centers[0]);
    free(new_centers);
    free(new_centers_len);

    return clusters;
}

